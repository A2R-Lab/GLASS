#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdint>
#include <hip/hip_cooperative_groups.h>
#include "../glass.cuh"
#include "./global_glass.cuh"
#include "gtest/gtest.h"

__host__
void printArray(int n, int * arr){
	std::cout<<"{ " << arr[0]; 
	for(int i = 1; i < n; i++){
		std::cout<<", "<<arr[i];
	}
	std::cout<<"}"<<std::endl;
}

class DeviceTest : public ::testing::Test{

        protected:
                void SetUp() override {
                        n = 10;
                        h_a = new int[n];
                        h_b = new int[n];
			h_c = new int;
                        for(int i = 0; i < n; i++){
                                h_a[i] = i;
                                h_b[i] = 2 * i;
                        }
			hipMalloc(&d_a, n * sizeof(int));
    			hipMalloc(&d_b, n * sizeof(int));
    			hipMalloc(&d_c, sizeof(int));
                	hipMemcpy(d_a, h_a, n * sizeof(int), hipMemcpyHostToDevice);
    			hipMemcpy(d_b, h_b, n * sizeof(int), hipMemcpyHostToDevice);
    			hipDeviceSynchronize();
		}
		void TearDown() override {
     			// Code here will be called immediately after each test (right
     			// before the destructor).
  			hipFree(d_a);
			hipFree(d_b);
			hipFree(d_c);
			delete h_a;
			delete h_b;
			delete h_c;
		}

        int n;
        int * h_a;
        int * h_b;
	int * h_c;
	int * d_a, *d_b, *d_c;
};

TEST_F(DeviceTest, DotProduct){
	global_dot<<<1, n>>>(d_c, n, d_a, d_b);
	hipDeviceSynchronize();

    	// copy the memory back
    	hipMemcpy(h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);
    	hipDeviceSynchronize();

	EXPECT_EQ(*h_c, 570);
	
}

int main(){
        ::testing::InitGoogleTest();
        return RUN_ALL_TESTS();
}




