#include "hip/hip_runtime.h"
#include <cpqp.cuh>
#include <hip/hip_runtime.h>
#include <iostream>

#define FORWARDPASS_THREADS 32


/*
This is a real QP that we solve in forward pass

P_test
array([[12.6191,  0.0136, -0.3856,  0.1301,  0.0613,  0.0454,  0.0173],
       [ 0.0136, 12.5569,  0.0865,  0.1415, -0.0619,  0.0701, -0.004 ],
       [-0.3856,  0.0865, 14.13  , -0.1305, -0.6172, -0.1599, -0.0509],
       [ 0.1301,  0.1415, -0.1305, 12.9557, -0.0594,  0.3747,  0.0042],
       [ 0.0613, -0.0619, -0.6172, -0.0594, 15.9793,  0.0845, -0.1174],
       [ 0.0454,  0.0701, -0.1599,  0.3747,  0.0845, 15.9542, -0.1119],
       [ 0.0173, -0.004 , -0.0509,  0.0042, -0.1174, -0.1119, 16.5841]])

q
array([-0.3501,  0.0418,  1.1519, -0.2435,  0.0982, -0.5519,  0.3218])

A_
array([[1., 0., 0., 0., 0., 0., 0.],
       [0., 1., 0., 0., 0., 0., 0.],
       [0., 0., 1., 0., 0., 0., 0.],
       [0., 0., 0., 1., 0., 0., 0.],
       [0., 0., 0., 0., 1., 0., 0.],
       [0., 0., 0., 0., 0., 1., 0.],
       [0., 0., 0., 0., 0., 0., 1.]])

lb
array([-4.9239, -4.9239, -4.9239, -4.9239, -4.9239, -4.9239, -4.9239])

ub
array([4.9239, 4.9239, 4.9239, 4.9239, 4.9239, 4.9239, 4.9239])
*/

template <typename T>
__global__ void test_cpqp(std::uint32_t dim, T *P, T *q, T *A, T *lb, T *ub, T *tmp1, T *res, T *tmp3, T *tmp4, T *tmp5,
                          T *tmp6, T *x_0, T *s_tmp, T *obj_tmp1, T *obj_tmp2, T *obj_res, T *x_new, T *dot_grad)
{
    cpqp<T>(dim, P, q, A, lb, ub, x_0, tmp1, res, tmp3, tmp4, tmp5, tmp6, s_tmp, obj_tmp1, obj_tmp2, obj_res, x_new,
            dot_grad);
    __syncthreads();
}

void cpqp_test_1()
{
    std::uint32_t num_control_dims = 7;

    double P[num_control_dims * num_control_dims] = {
        13.022784015920074, 0.1602830744821,     -0.975087407086252, 0.719470814817327,  0.269016915363258,
        0.174968209396238,  0.007070640863051,   0.160283074489815,  12.942923517222942, 0.312299783306563,
        0.965498329766035,  -0.231794957423362,  0.14404541886428,   0.270994416522598,  -0.975087407036581,
        0.312299783326669,  17.424384018629073,  -0.245860105289312, -3.189800608997873, -0.889226479786205,
        2.516164062142545,  0.719470814835794,   0.965498329758143,  -0.245860105387552, 15.562910964201624,
        -0.108626275482834, 1.080643826931862,   0.518484978850113,  0.269016914920422,  -0.231794957300408,
        -3.189800607052198, -0.108626275633404,  32.41501689914927,  1.365188996242705,  -24.624232216732917,
        0.17496820944846,   0.144045418847257,   -0.889226480018947, 1.080643826944995,  1.365188996237579,
        24.545707872269453, -0.918225780237453,  0.0070706420625,    0.270994415507298,  2.516164052600129,
        0.51848497757645,   -24.624232178925777, -0.918225785793746, 100.23596981903623};

    double q[num_control_dims] = {-4.959635017022945, 1.501537023540699,  1.465996601597597, -2.950586284512987,
                                  1.0058699524367,    -1.770490656955261, 5.591515635147498};

    double A[num_control_dims * num_control_dims] = {1.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 1.0, 0.0, 0.0, 0.0, 0.0,
                                                     0.0, 0.0, 0.0, 1.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 1.0, 0.0,
                                                     0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 1.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0,
                                                     0.0, 1.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 1.0};

    double lb[num_control_dims] = {-45, -45, -45, -45, -45, -45, -45};
    double ub[num_control_dims] = {45, 45, 45, 45, 45, 45, 45};
    double x_0[num_control_dims] = {0, 0, 0, 0, 0, 0, 0};

    double *d_P, *d_q, *d_A, *d_lb, *d_ub, *d_tmp1, *d_res, *d_tmp3, *d_tmp4, *d_tmp5, *d_tmp6, *d_x_0;
    double *d_s_tmp, *d_obj_tmp1, *d_obj_tmp2, *d_obj_res;
    double *d_x_new, *d_dot_grad;

    hipMalloc(&d_P, num_control_dims * num_control_dims * sizeof(double));
    hipMalloc(&d_q, num_control_dims * sizeof(double));
    hipMalloc(&d_A, num_control_dims * num_control_dims * sizeof(double));
    hipMalloc(&d_lb, num_control_dims * sizeof(double));
    hipMalloc(&d_ub, num_control_dims * sizeof(double));
    hipMalloc(&d_tmp1, num_control_dims * sizeof(double));
    hipMalloc(&d_res, num_control_dims * sizeof(double));
    hipMalloc(&d_tmp3, num_control_dims * sizeof(double));
    hipMalloc(&d_tmp4, num_control_dims * sizeof(double));
    hipMalloc(&d_tmp5, num_control_dims * sizeof(double));
    hipMalloc(&d_tmp6, num_control_dims * sizeof(double));
    hipMalloc(&d_x_0, num_control_dims * sizeof(double));
    hipMalloc(&d_res, num_control_dims * sizeof(double));
    hipMalloc(&d_s_tmp, FORWARDPASS_THREADS * sizeof(double));
    hipMalloc(&d_obj_tmp1, num_control_dims * sizeof(double));
    hipMalloc(&d_obj_tmp2, num_control_dims * sizeof(double));
    hipMalloc(&d_obj_res, num_control_dims * sizeof(double));
    hipMalloc(&d_x_new, num_control_dims * sizeof(double));
    hipMalloc(&d_dot_grad, num_control_dims * sizeof(double));

    hipMemcpy(d_P, P, num_control_dims * num_control_dims * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_q, q, num_control_dims * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_A, A, num_control_dims * num_control_dims * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_lb, lb, num_control_dims * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_ub, ub, num_control_dims * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_x_0, x_0, num_control_dims * sizeof(double), hipMemcpyHostToDevice);

    dim3 blockSize(1);
    dim3 gridSize(1);

    test_cpqp<<<gridSize, blockSize, FORWARDPASS_THREADS>>>(num_control_dims, d_P, d_q, d_A, d_lb, d_ub, d_tmp1, d_res,
                                                            d_tmp3, d_tmp4, d_tmp5, d_tmp6, d_x_0, d_s_tmp, d_obj_tmp1,
                                                            d_obj_tmp2, d_obj_res, d_x_new, d_dot_grad);
    hipDeviceSynchronize();

    double h_res[num_control_dims];
    hipMemcpy(h_res, d_res, num_control_dims * sizeof(double), hipMemcpyDeviceToHost);
    for (std::uint32_t i = 0; i < num_control_dims; i++)
        printf("%f ", h_res[i]);
    printf("\n");

    hipFree(d_P);
    hipFree(d_q);
    hipFree(d_A);
    hipFree(d_lb);
    hipFree(d_ub);
    hipFree(d_tmp1);
    hipFree(d_res);
    hipFree(d_tmp3);
    hipFree(d_tmp4);
    hipFree(d_tmp5);
    hipFree(d_tmp6);
    hipFree(d_x_0);
}

int main()
{
    cpqp_test_1();
    return 0;
}